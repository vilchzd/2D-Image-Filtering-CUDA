#include <iostream>   
#include <cstdlib>   
#include <hip/hip_runtime.h>

using namespace std;

__global__ void matrix(float *in, float *out, int width, int height) {
    extern __shared__ float mat[];
    int x = threadIdx.x;
    int y = threadIdx.y;
    mat[x * width + y] = in[x * width + y]; 
    __syncthreads();
    mat[x * width + y] /= 10; 
    out[x * width + y] = mat[x * width + y]; 
}

int main() {
    const int width = 5, height = 5;
    float *d_in, *d_out; 
    float *h_in = (float*)malloc(height*width*sizeof(float));
    float *h_out = (float*)malloc(height*width*sizeof(float));
    hipMalloc((void**)&d_in,width*height*sizeof(float));
    hipMalloc((void**)&d_out,width*height*sizeof(float));
    
    for (int i = 0; i < height*width; i++) {
        h_in[i] = i*10+10;
    }

    hipMemcpy(d_in, h_in, width*height*sizeof(float), hipMemcpyHostToDevice);
    
    dim3 grid_size(1);
    dim3 block_size(width,height);
    size_t shm_size = width * height * sizeof(float);
    matrix<<<grid_size,block_size, shm_size>>>(d_in, d_out, 5 , 5);
    
    hipMemcpy(h_out, d_out, width*height*sizeof(float), hipMemcpyDeviceToHost);
    
    for (int y = 0; y < height; y++) {
        for (int i = 0; i < width; i++) {
            cout << h_out[y*width + i] << " ";
        }
        cout << endl;
    }
    free(h_in);
    free(h_out);
    hipFree(d_in);
    hipFree(d_out);
    
    return 0;
}

/* __global__ void read_m(float *in, float *out, int w, int h) {
    const int height = 5;
    const int width = 5;
    __shared__ float matrix[height][width];

    int y = threadIdx.y;
    int x = threadIdx.x;
    if ()
    

    if (i < width && j < width) {
        matrix[threadIdx.y][threadIdx.x] = in[j * width + i];
    }

    __syncthreads();
    if (trans_i < width && trans_j < width) {
        out[trans_j * width + trans_i] = matrix[threadIdx.x][threadIdx.y];
    }
}
*/